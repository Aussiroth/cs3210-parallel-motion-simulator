#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <vector> 
#include <algorithm>
#include <chrono>
#include <random>
#include <atomic>
#include <stdio.h>

#define NUM_STREAMS 2

using namespace std;

mt19937 rng;
random_device rd;
__managed__ int n, l, r, s;

class Particle
{ 
	public: 
		operator string() const { 
			char buffer [200];
			snprintf(buffer, 200, "%d %.8lf %.8lf %.8lf %.8lf %d %d", i, x, y, vX, vY, pColl, wColl); 
			return buffer;
		}

		int i;
		double x;
		double y;
		double vX;
		double vY; 

		int pColl;
		int wColl;

		Particle() {};

		Particle(int i, double x, double y, double vX, double vY, int l) 
		{
			this -> i = i;
			this -> x = x;
			this -> y = y;
			this -> vX = vX;
			this -> vY = vY;
			this -> pColl = 0;
			this -> wColl = 0;
		}

		__device__ __host__ int getIndex()
		{
			return this->i;
		}
}; 


class JaggedMatrix
{
	public:
		int length;
		double **matrix;

		JaggedMatrix(int i) 
		{
			this->length = i;
			matrix = (double**) calloc(i, sizeof(double *));
			for (int k = 0; k < i; ++k) 
			{
				matrix[k] = (double *) calloc(k+1, sizeof(double));
			}
		}

		int get(int i, int j)
		{
			if (i < j) 
			{
				return matrix[j][i];
			}
			return matrix[i][j];
		}

		void set(int i, int j, double value)
		{
			if (i < j) 
			{
				matrix[j][i] = value;
			} else {
				matrix[i][j] = value;
			}

		}

		void destroy()
		{
			for (int k = 0; k < length; ++k)
			{
				free(matrix[k]);
			}
			free(matrix);
		}
};

class CollisionEvent 
{
	bool operator < (CollisionEvent other)
	{
		if (this->time == other.getTime()) return this->getSmallestIndex() > other.getSmallestIndex(); 
		return this->time > other.getTime();
	}

	public:
	Particle* first;
	double time;

	CollisionEvent() {}

	virtual ~CollisionEvent() {}

	CollisionEvent(Particle* first, double time)
	{
		this->first = first;
		this->time = time;
	}

	virtual void execute() {};

	double getTime()
	{
		return this->time;
	}

	double getSmallestIndex()
	{
		return (*first).getIndex();
	}
};

class ParticleCollisionEvent: public CollisionEvent
{
	public:
		bool operator == (ParticleCollisionEvent other)
		{
			int firstIndex = (*this->first).getIndex();
			int secondIndex = (*this->second).getIndex();
			int otherFirstIndex = (*other.first).getIndex();
			int otherSecondIndex = (*other.second).getIndex();
			return (firstIndex == otherSecondIndex && secondIndex == otherFirstIndex) ||
				(firstIndex == otherFirstIndex && secondIndex == otherSecondIndex);
		}
		Particle* second;


		ParticleCollisionEvent(Particle* first, Particle* second, double time)
			: CollisionEvent(first, time)
		{
			this->second = second;
		}

		void execute() 
		{
			if (first->getIndex() >= second->getIndex())
				return;
			//move them to proper position first
			first->x += time * first->vX;
			first->y += time * first->vY;
			second->x += time * second->vX;
			second->y += time * second->vY;

			//perform collision here
			//find normal vector
			double normalX = first->x - second->x;
			double normalY = first->y - second->y;
			double normalMag = sqrt(pow(normalX, 2) + pow(normalY, 2));
			normalX = normalX/normalMag; normalY = normalY/normalMag;
			double tangentX = -normalY;
			double tangentY = normalX;

			//compute velocity vectors wrt to normal and tangent
			double vFirstNormal = normalX * first->vX + normalY * first->vY;
			double vFirstTangent = tangentX * first->vX + tangentY * first->vY;
			double vSecondNormal = normalX * second->vX + normalY * second->vY;
			double vSecondTangent = tangentX * second->vX + tangentY * second->vY;

			//collision simply swaps velocities
			double temp = vFirstNormal;
			vFirstNormal = vSecondNormal;
			vSecondNormal = temp;

			first->vX = vFirstNormal * normalX + vFirstTangent * tangentX;
			first->vY = vFirstNormal * normalY + vFirstTangent * tangentY;
			second->vX = vSecondNormal * normalX + vSecondTangent * tangentX;
			second->vY = vSecondNormal * normalY + vSecondTangent * tangentY;

			//eliminate negative 0s
			if (first->vX == -0.0) first->vX = 0.0;
			if (first->vY == -0.0) first->vY = 0.0;
			if (second->vX == -0.0) second->vX = 0.0;
			if (second->vY == -0.0) second->vY = 0.0;

			//Continue to move them here
			//Check for wall collisions and stop the particle at wall if so
			double timeToMove;
			double xCollide = first->vX < 0 ? (first->x-r)/(0-first->vX) : ((double)l-r-first->x)/first->vX;
			double yCollide = first->vY < 0 ? (first->y-r)/(0-first->vY) : ((double)l-r-first->y)/first->vY;
			if (xCollide >= 1-time && yCollide >= 1-time) 
			{
				timeToMove = 1-time;
			}
			else
			{
				timeToMove = min(xCollide, yCollide);
			}
			first->x += timeToMove * first->vX;
			first->y += timeToMove * first->vY;
			first->pColl++;
			second->pColl++;
		}

		double getSmallestIndex()
		{
			return (*first).getIndex() < (*second).getIndex() ? (*first).getIndex() : (*second).getIndex();
		}
};

class WallCollisionEvent: public CollisionEvent
{
	public:

		WallCollisionEvent(Particle* first, double time)
			: CollisionEvent(first, time){}

		void execute() {
			//check for x wall collisions
			//check for y wall collisions
			double xCollide = first->vX < 0 ? (first->x-r)/(0-first->vX) : ((double)l-first->x-r)/first->vX;
			double yCollide = first->vY < 0 ? (first->y-r)/(0-first->vY) : ((double)l-first->y-r)/first->vY;
			int earlierTime = min(xCollide, yCollide);
			int laterTime = max(xCollide, yCollide);
			first->x += earlierTime * first->vX;
			first->y += earlierTime * first->vY;
			//Reverse direction depending on which collision happens first
			if (xCollide <= yCollide) {
				first->vX = -first->vX;
			}	
			if (yCollide <= xCollide) {
				first->vY = -first->vY;
			}
			//artificially set timing to allow particle to continue after hitting corner
			if (xCollide == yCollide) {
				laterTime = 1;
			}
			first->x += (min(1, laterTime)-earlierTime) * first->vX;	
			first->y += (min(1, laterTime)-earlierTime) * first->vY;
			first->wColl++;
		}
};

class NoCollisionEvent: public CollisionEvent
{
	public:

		NoCollisionEvent(Particle* first)
			: CollisionEvent(first, 1.0)
		{}

		void execute() {
			//simply move the particle
			first->x += first->vX;
			first->y += first->vY;
		}
};

__managed__ double** particleCollisionTimes;
__managed__ double* wallCollisionTimes;
__managed__ Particle* particles;
__managed__ CollisionEvent* found;
__managed__ CollisionEvent* temp;
hipStream_t streams[NUM_STREAMS];

__host__ void moveParticles(Particle* particles);
__global__ void timeParticleCollision();
__global__ void timeWallCollision();
__global__ void executeParticleCollision();
__global__ void executeWallCollision();
__global__ void executeNoCollision();

__host__ int main (void)
{
	string command; // simulator command
	cin >> n >> l >> r >> s >> command;

	rng.seed(rd());
	uniform_real_distribution<double> pos(r, l-r);
	uniform_real_distribution<double> velocity((double)l/(8*r), (double)l/4);
	//vector<Particle*> particles; 
	hipError_t cudaStatus = hipMallocManaged(&particles, sizeof(Particle) * n);
	cout << cudaStatus << endl;
	int scanned;
	for (scanned = 0; scanned < n; ++scanned)
	{
		int index; 
		double x;
		double y;
		double vX;
		double vY; 
		int count;
		count = scanf("%d %lf %lf %lf %lf", &index, &x, &y, &vX, &vY);
		if (count == EOF || count <= 0) break;

		particles[scanned] = Particle(index, x, y, vX, vY, l);
	}
	for (int j = scanned; j < n; j++)
	{
		double x = pos(rng);
		double y = pos(rng);
		double vX = velocity(rng);
		double vY = velocity(rng);
		particles[scanned] = Particle(j, x, y, vX, vY, l);
	}


	for (int i = 0; i < NUM_STREAMS; ++i) 
	{
		hipStreamCreate(&streams[i]);
	}

	auto start = chrono::high_resolution_clock::now();

	for (int i = 0; i < s; ++i)
	{	
		moveParticles(particles);
		if (!command.compare("print"))
		{
			for (int j = 0; j < n; ++j)
			{
				cout << i << " " << (string) particles[j] << endl;
			}
		}
	}

	auto finish = std::chrono::high_resolution_clock::now();

	for (int j = 0; j < n; ++j)
	{
		cout << (string) particles[j] << endl;
	}
	double timeTaken = (double)chrono::duration_cast<chrono::nanoseconds>(finish-start).count()/1000000000;
	// printf("Time taken: %.5f s\n", timeTaken);

	return 0;
}


__host__ void moveParticles(Particle* particles) 
{
	hipMallocManaged(&particleCollisionTimes, sizeof(double*) * n);
	for (int i = 0; i < n; ++i)
	{
		hipMallocManaged(&particleCollisionTimes[i], sizeof(double) * n);
	}
	hipMallocManaged(&wallCollisionTimes, sizeof(double) * n);

	hipMallocManaged(&found, sizeof(CollisionEvent) * n);
	hipMallocManaged(&temp, sizeof(CollisionEvent) * n);

	for (int i = 0; i < n; ++i)
	{
		found[n] = (CollisionEvent) NULL;
		temp[n] = (CollisionEvent) NULL;
	}
	// time of particle-particle collisions
	// JaggedMatrix particleCollisionTimes = JaggedMatrix(n);
	// time of particle-wall collisions
	// double wallCollisionTimes[n] = {};

	// calculate collision times
	timeWallCollision<<<(n-1)/64+1, 64, 0, streams[0]>>>();
	dim3 threadsPerBlock(16, 16, 1);
	dim3 blocksPerGrid((n-1)/16 + 1, (n-1)/16 + 1);

	timeParticleCollision<<<blocksPerGrid, threadsPerBlock, 0, streams[1]>>>();

	hipDeviceSynchronize();
	for (int i = 0; i < n; ++i) {
		for (int j = 0; j < n; ++j) {
			cout<< particleCollisionTimes[i][j] << " ";
		}
		cout << endl;
	}

	for (int j = 0; j < n; ++j) {
		cout<< wallCollisionTimes[j] << " ";
	}
	cout << endl;

	int foundCount = 0;
	while (foundCount != n)
	{   
		// CollisionEvent* temp[n];
		// #pragma omp parallel for
		for (int i = 0; i < n; ++i)
		{   
			// first assume no collision
			temp[i] = new NoCollisionEvent(&particles[i]);

			// check for particle-wall collision
			if (wallCollisionTimes[i] < (*temp[i]).getTime() && wallCollisionTimes[i] < 1)
			{
				temp[i] = new WallCollisionEvent(&particles[i], wallCollisionTimes[i]);
			}

			// check for particle-particle collision
			for (int j = 0; j < n; ++j)
			{
				if (i == j) continue;

				double time = particleCollisionTimes[i][j];
				if (time > -1 && time < (*temp[i]).getTime() && time < 1 && found[j] == NULL) {
					temp[i] = new ParticleCollisionEvent(&particles[i], &particles[j], time);
				}
			}
		}

		for (int i = 0; i < n; ++i)
		{
			if (found[i] != NULL) continue;

			CollisionEvent* e = temp[i];

			// particle-particle collision
			if(ParticleCollisionEvent* v = dynamic_cast<ParticleCollisionEvent*>(e))
			{

				int otherIndex = (*(*v).second).getIndex();
				if (ParticleCollisionEvent* v2 = dynamic_cast<ParticleCollisionEvent*>(temp[otherIndex]))
				{
					if (*v == *v2) 
					{
						found[i] = temp[i];
						++foundCount;
					}
				}

			}
			// particle-wall collision or no collision
			else
			{
				found[i] = temp[i];
				++foundCount;
			}
		}
	}

	executeParticleCollision<<<(particleCollisionsCount-1)/64+1, 64, 0, streams[0]>>>();
	executeWallCollision<<<(wallCollisionsCount-1)/64+1, 64, 0, streams[1]>>>();
	executeNoCollision<<<(n-1)/64+1, 64, 0, streams[1]>>>(); 
	/*for (int i = 0; i < n; ++i)
	  {
	  (*found[i]).execute();
	  }*/
}

// input: 2 Particles
// output: Returns time taken before collision occurs if they collide, negative value otherwise.
__global__ void timeParticleCollision()
{
	int firstIndex = blockIdx.x * blockDim.x + threadIdx.x;
	int secondIndex = blockIdx.y * blockDim.y + threadIdx.y;
	if (firstIndex >= n || secondIndex >= n || firstIndex >= secondIndex) return;
	else
	{
		Particle first = particles[firstIndex];
		Particle second = particles[secondIndex];
		//a, b and c are as in the quadratic formula representation.
		//t, the time taken for the 2 circles to touch, is the unknown variable we are solving for
		//by taking difference in circle centres, setting an unknown t for collision time, and then taking distance moved in time t,
		//we can solve for t such that the circle centers are <= 2r and therefore collide. 4r^2 is to solve for radius distance.
		double c = pow((first.x-second.x), 2) + pow((first.y - second.y), 2) - 4*r*r;
		double b = 2*((first.x - second.x)*(first.vX - second.vX) + (first.y - second.y)*(first.vY-second.vY));
		double a = pow((first.vX-second.vX), 2) + pow((first.vY - second.vY), 2);

		//check for solution
		double solfirst;
		if (b*b-4*a*c < 0)
		{
			solfirst = 100000.0;
		} 
		else
		{
			//else if there is a solution, the one with smaller value should be the main collision. Second value is after the 2 circles phase through each other
			solfirst = (-sqrt(b*b-4*a*c)-b)/(2*a);
			solfirst = solfirst < 0 ? 0 : solfirst;
		}
		printf("solfirst: %lf\n", solfirst);
		particleCollisionTimes[first.i][second.i] = solfirst;
		particleCollisionTimes[second.i][first.i] = solfirst;
	}
}

// input: 1 Particle
// output: Returns time taken before collision occurs if it collides with wall, negative value otherwise.
__global__ void timeWallCollision()
{
	int particleIndex = blockIdx.x * blockDim.x + threadIdx.x;
	if (particleIndex >=n) {
		return;
	}
	else
	{
		Particle particle = particles[particleIndex];
		//check for x wall, y wall collisions
		double xCollide = particle.vX < 0 ? (particle.x-r)/(0-particle.vX) : ((double)l-particle.x-r)/particle.vX;
		double yCollide = particle.vY < 0 ? (particle.y-r)/(0-particle.vY) : ((double)l-particle.y-r)/particle.vY;
		wallCollisionTimes[particle.i] = min(xCollide, yCollide);
	}
}

__global__ void executeParticleCollision()
{
	int particleIndex = blockIdx.x * blockDim.x + threadIdx.x;
	if (particleIndex < particleCollisionsCount)
	{
		particleCollisions[particleIndex].execute();
	}
}

__global__ void executeWallCollision()
{
	int particleIndex = blockIdx.x * blockDim.x + threadIdx.x;
	if (particleIndex < wallCollisionsCount)
	{
		wallCollisions[particleIndex].execute();
	}
}

__global__ void executeNoCollision()
{
	int particleIndex = blockIdx.x * blockDim.x + threadIdx.x;
	if (particleIndex < noCollisionsCount)
	{
		noCollisions[particleIndex].execute();
	}
}
